#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
// #include <stdlib.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void test(char *A, char *B, int n) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= n) return;
    B[index] = A[index] == '0' ? '$' : A[index];
}

int main () {
    int n = 5;
	thrust::host_vector<char> h_A(n + 1);
    thrust::host_vector<char> h_B(n + 1);
	thrust::device_vector<char> d_A;
    thrust::device_vector<char> d_B;

    for (int i = 0; i < n; i++) {
        h_A[i] = '0' + i;
    }

	d_A = h_A;
    d_B = h_B;

    dim3 block(32, 1);
    dim3 grid((n + block.x - 1) / block.x, 1);
	char *pd_A = thrust::raw_pointer_cast(&d_A[0]);
	char *pd_B = thrust::raw_pointer_cast(&d_B[0]);

	test<<< grid, block >>>(pd_A, pd_B, n);

    h_B = d_B;

    for (int i = 0; i < n; i++) {
        printf("%c ", h_B[i]);
    }

}