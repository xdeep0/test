#include "hip/hip_runtime.h"
#include <stdio.h>
// #include <string.h>
// #include <stdlib.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

// (char* T, char* BWT, int* SA, int n) {
__global__ void test(char *A, char *B, int *C,int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n) return;
    B[i] = C[i] == 0 ? '$' : A[C[i] - 1];
}

int main () {
    int n = 12;
	thrust::host_vector<char> h_A(n);
	thrust::device_vector<char> d_A;

    char *T = (char *)malloc((n + 1) * sizeof(char));
    for (int i = 0; i < n; i++) {
        T[i] = "mississippi$"[i];
    }

    int SA_tmp[] = {11,10,7,4,1,0,9,8,6,3,5,2};
    int *SA = (int *)malloc(n * sizeof(int));
    for (int i = 0; i < n; i++) {
        SA[i] = SA_tmp[i];
    }

	d_A = h_A;

    dim3 block(32, 1);
    dim3 grid((n + block.x - 1) / block.x, 1);
	char *pd_A = thrust::raw_pointer_cast(&d_A[0]);

	test<<< grid, block >>>(T, pd_A, SA, n);

    h_A = d_A;

    for (int i = 0; i < n; i++) {
        printf("%c ", h_A[i]);
    }

}