#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
// #include <stdlib.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// (char* T, char* BWT, int* SA, int n) {
__global__ void test(char *A, char *B, int *C,int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n) return;
    B[i] = C[i] == 0 ? '$' : A[C[i] - 1];
    // B[index] = A[index] == '0' ? '$' : A[index];
}

int main () {
    int n = 12;
	thrust::host_vector<char> h_A(n + 1);
    thrust::host_vector<char> h_B(n + 1);
	thrust::device_vector<char> d_A;
    thrust::device_vector<char> d_B;

    char *T = (char *)malloc((n + 1) * sizeof(char));
    for (int i = 0; i < n; i++) {
        T[i] = "mississippi$"[i];
    }

    int SA_tmp[] = {11,10,7,4,1,0,9,8,6,3,5,2};
    int *SA = (int *)malloc(n * sizeof(int));
    for (int i = 0; i < n; i++) {
        SA[i] = SA_tmp[i];
    }


    // for (int i = 0; i < n; i++) {
    //     h_A[i] = '0' + i;
    // }

	d_A = h_A;
    d_B = h_B;

    dim3 block(32, 1);
    dim3 grid((n + block.x - 1) / block.x, 1);
	char *pd_A = thrust::raw_pointer_cast(&d_A[0]);
	char *pd_B = thrust::raw_pointer_cast(&d_B[0]);

	test<<< grid, block >>>(T, pd_B, SA, n);

    h_B = d_B;

    for (int i = 0; i < n; i++) {
        printf("%c ", h_B[i]);
    }

}