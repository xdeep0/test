#include "hip/hip_runtime.h"
// #include <stdio.h>
// // #include <string.h>
// // #include <stdlib.h>
// #include <thrust/host_vector.h>
// #include <thrust/device_vector.h>
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

// // (char* T, char* BWT, int* SA, int n) {
// __global__ void test(char *A, char *B, int *C,int n) {
// 	int i = blockIdx.x * blockDim.x + threadIdx.x;
// 	if (i >= n) return;
//     B[i] = C[i] == 0 ? '$' : A[C[i] - 1];
// }

// int main () {
//     const int n = 12;
//     char A[] = "mississippi$";
// 	thrust::host_vector<char> h_B(n);
// 	thrust::device_vector<char> d_B = h_B;

//     // char *T = (char *)malloc((n + 1) * sizeof(char));
//     // for (int i = 0; i < n; i++) {
//     //     T[i] = "mississippi$"[i];
//     // }
//     // char T[] = "mississippi$";

//     // int SA_tmp[] = {11,10,7,4,1,0,9,8,6,3,5,2};
//     // int *SA = (int *)malloc(n * sizeof(int));
//     // for (int i = 0; i < n; i++) {
//     //     SA[i] = SA_tmp[i];
//     // }
//     // int SA[] = {11,10,7,4,1,0,9,8,6,3,5,2};
//     int C[] = {11,10,7,4,1,0,9,8,6,3,5,2};

// 	// d_B = h_B;

//     dim3 block(8, 1);
//     dim3 grid((n + block.x - 1) / block.x, 1);
// 	char *pd_B = thrust::raw_pointer_cast(&d_B[0]);

// 	// test<<< grid, block >>>(T, pd_B, SA, n);
//     test<<< grid, block >>>(A, pd_B, C, n);

//     h_B = d_B;

//     for (int i = 0; i < n; i++) {
//         printf("%c ", h_B[i]);
//     }

// }
#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void test(char* A, char* B, int* C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    B[i] = C[i] != 0 ? A[C[i] - 1] : '$';
}

int main() {
    const int n = 12;
    char A[] = "mississippi$";
    int C[] = { 11,10,7,4,1,0,9,8,6,3,5,2 };

    thrust::device_vector<char> d_A(A, A+n);
    thrust::device_vector<char> d_B(n);
    thrust::device_vector<int>  d_C(C, C+n);

    dim3 block(8, 1);
    dim3 grid((n + block.x - 1) / block.x, 1);
    test<<<grid, block >>>(
        thrust::raw_pointer_cast(&d_A[0]),
        thrust::raw_pointer_cast(&d_B[0]),
        thrust::raw_pointer_cast(&d_C[0]),
        n);

    thrust::host_vector<char> h_B = d_B;

    for (int i = 0; i < n; i++) {
        printf("%c ", h_B[i]);
    }
}
