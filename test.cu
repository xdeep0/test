#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
// #include <stdlib.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void test(int *A, int *B, int n) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= n) return;
    B[index] = A[index];
}

int main () {
    int n = 5;
	thrust::host_vector<int> h_A(n + 1);
    thrust::host_vector<int> h_B(n + 1);
	thrust::device_vector<int> d_A;
    thrust::device_vector<int> d_B;

    for (int i = 0; i < n; i++) {
        h_A[i] = i;
    }

	d_A = h_A;
    d_B = h_B;

    dim3 block(32, 1);
    dim3 grid((n + block.x - 1) / block.x, 1);
	int *pd_A = thrust::raw_pointer_cast(&d_A[0]);
	int *pd_B = thrust::raw_pointer_cast(&d_B[0]);

	test<<< grid, block >>>(pd_A, pd_B, n);

    h_B = d_B;

    for (int i = 0; i < n; i++) {
        printf("%d ", h_B[i]);
    }

}